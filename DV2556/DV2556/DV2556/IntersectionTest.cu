#include "IntersectionTest.h"

std::vector<IntersectionResult> IntersectionTest::runTest(TestData* data)
{
	std::vector<IntersectionResult> resultVector(data->triangleCount);

	if (result != nullptr)
	{
		hipFree(result);
	}

	hipMalloc((void**) &result, data->triangleCount * sizeof(IntersectionResult));

	// start timer
	test(data);
	hipDeviceSynchronize();
	// end timer

	// collect results
	hipMemcpy(resultVector.data(), result, data->triangleCount * sizeof(IntersectionResult), hipMemcpyKind::hipMemcpyDeviceToHost);

	return resultVector;
}


IntersectionTest::~IntersectionTest()
{
	hipFree(result);
}