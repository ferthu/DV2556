#include "IntersectionTest.h"

TestResult IntersectionTest::runTest(TestData* data)
{
	TestResult retResult;
	retResult.intersectionResults.resize(data->triangleCount);
	//std::vector<IntersectionResult> resultVector(data->triangleCount);

	//if (result != nullptr)
	//{
	//	hipFree(result);
	//}
	
	if (result == nullptr)
		hipMalloc((void**) &result, data->triangleCount * sizeof(IntersectionResult));

	// start timer
	StopWatch sw;
	sw.start();

	test(data);
	hipDeviceSynchronize();
	// end timer
	retResult.duration = sw.getTimeInSeconds();
	// collect results
	hipMemcpy(retResult.intersectionResults.data(), result, data->triangleCount * sizeof(IntersectionResult), hipMemcpyKind::hipMemcpyDeviceToHost);


	return retResult;
}


IntersectionTest::~IntersectionTest()
{
	hipFree(result);
}