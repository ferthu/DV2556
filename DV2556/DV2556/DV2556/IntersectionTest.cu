#include "IntersectionTest.h"

std::vector<IntersectionResult> IntersectionTest::runTest(TestData* data)
{
	std::vector<IntersectionResult> resultVector;

	if (result != nullptr)
	{
		hipFree(result);
	}

	hipMalloc((void**) &result, data->triangleCount * sizeof(IntersectionResult));

	// start timer
	test(data);
	// end timer

	// collect results

	return resultVector;
}


IntersectionTest::~IntersectionTest()
{
	hipFree(result);
}