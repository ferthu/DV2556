#include "TestData.h"

TestData::TestData(float hitrate, size_t triangleCount)
{
	this->triangleCount = triangleCount;

	Triangle* cpuTriangles = new Triangle[triangleCount];
	// generate triangles and ray
	Ray* cpuRay = new Ray();
	cpuRay->origin = vec3(0.0f);
	cpuRay->direction = vec3(0.0f, 0.0f, 1.0f);

	// Allocate on GPU
	hipMalloc((void**) &triangles, triangleCount * sizeof(Triangle));
	hipMalloc((void**) &ray, sizeof(Ray));
	
	// Copy to GPU
	hipMemcpy(cpuTriangles, triangles, triangleCount * sizeof(Triangle), hipMemcpyHostToDevice);
	hipMemcpy(cpuRay, ray, sizeof(Ray), hipMemcpyHostToDevice);

	// Delete CPU allocations
	delete cpuTriangles;
	delete cpuRay;
}

TestData::~TestData()
{
	hipFree(ray);
	hipFree(triangles);
}