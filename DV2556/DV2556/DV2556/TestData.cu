#include "TestData.h"

TestData::TestData(float hitrate, size_t triangleCount)
{
	cpuTriangleCount = triangleCount;

	Triangle* cpuTriangles = new Triangle[triangleCount];
	// generate triangles and ray
	Ray* cpuRay = new Ray();
	cpuRay->origin = vec3(0.0f);
	cpuRay->direction = vec3(0.0f, 0.0f, 1.0f);

	// Allocate on GPU
	hipMalloc((void**) &triangles, triangleCount * sizeof(Triangle));
	hipMalloc((void**) &ray, sizeof(Ray));
	hipMalloc((void**) &(this->triangleCount), sizeof(size_t));
	
	// Copy to GPU
	hipMemcpy(cpuTriangles, triangles, triangleCount * sizeof(Triangle), hipMemcpyHostToDevice);
	hipMemcpy(cpuRay, ray, sizeof(Ray), hipMemcpyHostToDevice);
	hipMemcpy(&cpuTriangleCount, this->triangleCount, sizeof(size_t), hipMemcpyHostToDevice);

	// Delete CPU allocations
	delete cpuTriangles;
	delete cpuRay;
}

TestData::~TestData()
{
	hipFree(ray);
	hipFree(triangles);
	hipFree(triangleCount);
}